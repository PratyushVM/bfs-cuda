#include<stdio.h>
#include<utility>
#include<hip/hip_runtime.h>
#include<time.h>

class graph
{
public:

    int num_vertices, num_edges;
    int *depth;
    std::pair<int,int> *edgelist;

    graph(int v, int e)
    {
        num_vertices = v;
        num_edges = e;
        depth = (int*)malloc(v*sizeof(int));
        edgelist = (std::pair<int,int> *)malloc(e*sizeof(std::pair<int,int>));
    }
};

/*void readgraph(graph &g, int argc, char **argv)
{
    int v,e;

    bool fromstdin = (argc > 2);

    if(!fromstdin)
    {
        scanf("%d %d",&v,&e);
    }

    else
    {
        srand(0);
        v = atoi(argv[2]);
        e = atoi(argv[3]);
    }


    if(fromstdin)
    {
        int a,b;
        for(int i=0;i<e;i++)
        {
            scanf("%d %d",&a,&b);
            g.edgelist[i] = std::make_pair(a,b);
        }
    }

    else
    {
        int a,b;
        for(int i=0;i<e;i++)
        {
            a = rand() % v;
            b = rand() % v;
            g.edgelist[i] = std::make_pair(a,b);
        }
    }

}*/

__global__ void initialize(graph g, int start)
{
    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;

    if(id == start)
    {
        g.depth[id] = 0;
    }

    else
    {
        g.depth[id] = -1;
    }

}


__global__ void bfs(graph &g, int start, int current_depth, bool *stop)
{
    unsigned int e = blockIdx.x*blockDim.x + threadIdx.x;

    int v1 = g.edgelist[e].first;
    int v2 = g.edgelist[e].second;

    int d1 = g.depth[v1];
    int d2 = g.depth[v2];

    if(d1 != -1 && d2 == -1)
    {
        g.depth[v2] = current_depth + 1;
        *stop = false;
    }

    else if(d2 != -1 && d1 == -1)
    {
        g.depth[v1] = current_depth + 1;
        *stop = false;
    }

}

int main(int argc, char **argv)
{
    /*int v,e;
    bool fromstdin = (argc > 2);

    if(!fromstdin)
    {
        scanf("%d %d",&v,&e);
    }


    else
    {
        srand(0);
        v = atoi(argv[2]);
        e = atoi(argv[3]);
    }

    graph g(v,e);

    if(fromstdin)
    {
        int a,b;
        for(int i=0;i<e;i++)
        {
            scanf("%d %d",&a,&b);
            g.edgelist[i] = std::make_pair(a,b);
        }
    }

    else
    {
        int a,b;
        for(int i=0;i<e;i++)
        {
            a = rand() % v;
            b = rand() % v;
            g.edgelist[i] = std::make_pair(a,b);
        }
    }
    */
    //int *start = (int*)argv[1];
    int l = 0;
    int *start = (int *)2;
    //readgraph(g,argc,argv);
    
    graph g(5,4);
    for(int i=0;i<4;i++)
    {
        g.edgelist[i] = std::make_pair(i,i+1);
    }

    printf("Number of vertices : %d\n",g.num_vertices);
    printf("Number of edges : %d\n",g.num_edges);

    for(int i=0;i<g.num_vertices;i++)
    {
        g.depth[i] = -1;
    }
    g.depth[2] = 0;
    graph *gg;
    hipMalloc(&gg,sizeof(g));
    hipMemcpy(gg,&g,sizeof(g),hipMemcpyHostToDevice);

    //initialize<<<1,g.num_vertices>>>(*gg,*start);
    


    printf("BFS of graph :\n");
    bool done = false;
    bool *stop;

    hipMalloc(&stop,sizeof(bool));

    //while(!done)
    {
        done = true;
        bfs<<<1,g.num_edges>>>(*gg,2,l,&done);
        hipMemcpy(&done,stop,sizeof(done),hipMemcpyDeviceToHost);
        l++;
    }

    hipMemcpy(&g,gg,sizeof(g),hipMemcpyDeviceToHost);

    printf("Vertex - Level\n");

    for(int i=0; i<(g).num_vertices; i++)
    {
        printf("   %d       %d\n",i,(g).depth[i]);
    }

    return 0;
}