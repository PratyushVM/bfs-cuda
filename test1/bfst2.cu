#include<stdio.h>
#include<hip/hip_runtime.h>
#include<utility>

#define pii std::pair<int,int>
#define mp std::make_pair
#define f first
#define s second

//declaration of class object
class graph
{
public:
    
    int v,e; //number of vertices and edges
    int *depth; //array that stores depth (or) distance of each vertex from source
    pii *edgelist; //list of edges in the form of (vertex1,vertex2) pairs  

};

//Kernel that initializes depth array of graph

__global__ void init(graph *g, int start)
{
    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x; //id of thread

    //checking if vertex is the starting point of bfs or not, and initializing depth value respectively
    if(id == start)
    {
        g->depth[id] = 0;
    }

    else
    {
        g->depth[id] = -1;
    }

}

//Kernel invoked in bfs routine

__global__ void bfs(graph *g, bool *done)
{
    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x; //id of thread

    //storing corresponding vertices and depth values into thread's local memory 
    int v1 = g->edgelist[id].f;
    int v2 = g->edgelist[id].s;
    int d1 = g->depth[v1];
    int d2 = g->depth[v2];

    //checking if vertex of next depth value is discovered
    if(d1 != -1 && d2 == -1)
    {
        g->depth[v2] = g->depth[v1] + 1;
        *done = false;
    }
    
    else if(d2 != -1 && d1 == -1)
    {
        g->depth[v1] = g->depth[v2] + 1;
        *done = false;
    }

}

//main function

int main()
{
    //declaration of variables to store graph data on host and device
    graph *cpu_g,*gpu_g;
    int *c_depth,*g_depth;
    pii *c_edgelist,*g_edgelist;
    int nv = 5;
    int ne = 4;
    int start = 2;

    //allocating host memory for data of graph in host
    c_depth = (int*)malloc(nv*sizeof(int));
    c_edgelist = (pii *)malloc(ne*sizeof(pii));

    //allocating host memory for graph object
    cpu_g = (graph*)malloc(sizeof(graph));

    //assigning values to data members of graph object from host data
    cpu_g->v = nv;
    cpu_g->e = ne;
    cpu_g->depth = c_depth;
    cpu_g->edgelist = c_edgelist;
    
    for(int i=0; i<ne; i++)
    {
        c_edgelist[i] = mp(i,i+1);
    }

    //allocating device memory for graph object on GPU    
    hipMalloc((void**)&gpu_g,sizeof(graph));
    hipMalloc((void**)&g_depth,nv*sizeof(int));
    hipMalloc((void**)&g_edgelist,ne*sizeof(pii));

    //copying host data onto device
    hipMemcpy(g_depth,c_depth,nv*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(g_edgelist,c_edgelist,ne*sizeof(pii),hipMemcpyHostToDevice);
    hipMemcpy(gpu_g,cpu_g,sizeof(graph),hipMemcpyHostToDevice);

    hipMemcpy(&(gpu_g->edgelist),&g_edgelist,sizeof(pii *),hipMemcpyHostToDevice);
    hipMemcpy(&(gpu_g->depth),&g_depth,sizeof(int*),hipMemcpyHostToDevice);

    //invoking kernel to initialize the depth array of the graph 
    init<<<1,nv>>>(gpu_g,start);

    //declaration of bool variables in host for routine to invoke bfs kernel
    bool *cpu_done;
    cpu_done = (bool*)malloc(sizeof(bool));
    *cpu_done = false;

    //declaration of bool variables in device for routine to invoke bfs kernel   
    bool *gpu_done;
    hipMalloc((void**)&gpu_done,sizeof(bool));
    hipMemcpy(gpu_done,cpu_done,sizeof(bool),hipMemcpyHostToDevice);

    // routine that invokes bfs kernel from host

    while(!(*cpu_done))
    {
        *cpu_done = true;
        hipMemcpy(gpu_done,cpu_done,sizeof(bool),hipMemcpyHostToDevice);
        bfs<<<1,ne>>>(gpu_g,gpu_done);
        hipMemcpy(cpu_done,gpu_done,sizeof(bool),hipMemcpyDeviceToHost);
    }

    //copying device data back into host memory
    hipMemcpy(c_depth,g_depth,nv*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(c_edgelist,g_edgelist,ne*sizeof(pii),hipMemcpyDeviceToHost);
    hipMemcpy(cpu_g,gpu_g,sizeof(graph),hipMemcpyDeviceToHost);

    cpu_g->edgelist = c_edgelist;
    cpu_g->depth = c_depth;

    //printing depth of vertices from host memory
    for(int i=0;i<nv;i++)
    {
        printf("The depth of vertex %d is %d\n",i,cpu_g->depth[i]);
    }

}