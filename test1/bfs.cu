#include "hip/hip_runtime.h"
#include<stdio>
#include<cuda>
#include<utility>

#define pii std::pair<int,int>
#define mp std::make_pair
#define f first
#define s second

class graph
{
public:
    
    int v,e; //number of vertices and edges
    int *depth; //array that stores depth (or) distance of each vertex from source
    pii *edgelist; //list of edges in the form of (vertex1,vertex2) pairs  

    graph(int a,int b)
    {
        this->v = a;
        e = b;
        depth = (int*)malloc(v*sizeof(int));
        edgelist =  (pii*)malloc(e*sizeof(pii));
    }
};

__global__ void initialize(graph *g, int start)
{
    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;

    if(id == start)
    g->depth[id] = 0;

    else
    g->depth[id] = -1;
}


int main()
{
    //sample graph
    graph g(5,4);
    graph *cg,*gg;
    cg = &g;
    int start = 2;
    for(int i=0; i<4; i++)
    {
        g.edgelist[i] = mp(i,i+1);
    }

    hipMalloc((void**)&gg,sizeof(graph));
    hipMalloc((void**)&gg->edgelist,sizeof(cg->edgelist));
    hipMalloc((void**)&gg->depth,sizeof(cg->depth));

    hipMemcpy(gg,cg,sizeof(graph),hipMemcpyHostToDevice);
    hipMemcpy(gg->edgelist,cg->edgelist,sizeof(cg->edgelist),hipMemcpyHostToDevice);
    hipMemcpy(gg->depth,cg->depth,sizeof(cg->depth),hipMemcpyHostToDevice);

    initialize<<<1,g.v>>>(gg,start);

    hipMemcpy(cg,gg,sizeof(graph),hipMemcpyDeviceToHost);
    hipMemcpy(cg->edgelist,gg->edgelist,sizeof(gg->edgelist),hipMemcpyDeviceToHost);
    hipMemcpy(gg->depth,gg->depth,sizeof(gg->depth),hipMemcpyDeviceToHost);

    for(int i=0;i<5;i++)
    {
        printf("%d ",cg->depth[i]);
    }


    return 0;
}