#include<stdio.h>
#include<hip/hip_runtime.h>
#include<utility>

#define pii std::pair<int,int>
#define mp std::make_pair
#define f first
#define s second

//declaration of class object

class graph
{
public:
    
    int v,e; //number of vertices and edges
    int *depth; //array that stores depth (or) distance of each vertex from source
    pii *edgelist; //list of edges in the form of (vertex1,vertex2) pairs  

};

//Kernel that initializes depth array of graph

__global__ void init_depth_kernel(graph *g, int start)
{
    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x; //id of thread

    //checking if vertex is the starting point of bfs or not, and initializing depth value respectively
    if(id == start)
    {
        g->depth[id] = 0;
    }

    else
    {
        g->depth[id] = -1;
    }

}

//Kernel invoked in bfs routine

__global__ void bfs_kernel(graph *g, bool *done)
{
    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x; //id of thread

    //storing corresponding vertices and depth values into thread's local memory 
    int v1 = g->edgelist[id].f;
    int v2 = g->edgelist[id].s;
    int d1 = g->depth[v1];
    int d2 = g->depth[v2];

    //checking if vertex of next depth value is discovered
    if(d1 != -1 && d2 == -1)
    {
        g->depth[v2] = g->depth[v1] + 1;
        *done = false;
    }
    
    else if(d2 != -1 && d1 == -1)
    {
        g->depth[v1] = g->depth[v2] + 1;
        *done = false;
    }

}

//host function that invokes bfs_kernel iteratively 

void simple_bfs(graph *cpu_g, graph *gpu_g, bool *cpu_done, bool *gpu_done)
{
    while(!(*cpu_done))
    {
        *cpu_done = true;
        hipMemcpy(gpu_done,cpu_done,sizeof(bool),hipMemcpyHostToDevice);
        bfs_kernel<<<1,cpu_g->e>>>(gpu_g,gpu_done);
        hipMemcpy(cpu_done,gpu_done,sizeof(bool),hipMemcpyDeviceToHost);
    }

}

//host function that reads graph data from command line

void readgraph(pii *c_edgelist, int nv, int ne, int argc, char **argv)
{
    if(argc <= 4 || argc%2 != 0)
    {
        printf("Enter valid arguments in command line\n");
        exit(0);
    }

    else
    {
        int i,j;
        for(i=0, j=4;j<argc-1;i++,j+=2)
        {
            c_edgelist[i] = mp(atoi(argv[j]),atoi(argv[j+1]));
        }
    }
    
}

//host function to print the depth of each vertex

void printgraph(graph *cpu_g)
{
    for(int i=0;i<cpu_g->v;i++)
    {
        printf("The depth of vertex %d is %d\n",i,cpu_g->depth[i]);
    }
}

//main function

int main(int argc, char **argv)
{
    //declaration of variables to store graph data on host and device
    graph *cpu_g,*gpu_g;
    int *c_depth,*g_depth;
    pii *c_edgelist,*g_edgelist;

    int nv = atoi(argv[1]); //number of vertices
    int ne = atoi(argv[2]); //number of edges
    int start = atoi(argv[3]);  //starting vertex for bfs

    //allocating host memory for data of graph in host
    c_depth = (int*)malloc(nv*sizeof(int));
    c_edgelist = (pii *)malloc(ne*sizeof(pii));

    //invoking function to read graph data from command line
    readgraph(c_edgelist,nv,ne,argc,argv);

    //allocating host memory for graph object
    cpu_g = (graph*)malloc(sizeof(graph));

    //assigning values to data members of graph object from host data
    cpu_g->v = nv;
    cpu_g->e = ne;
    cpu_g->depth = c_depth;
    cpu_g->edgelist = c_edgelist;

    //allocating device memory for graph object on GPU    
    hipMalloc((void**)&gpu_g,sizeof(graph));
    hipMalloc((void**)&g_depth,nv*sizeof(int));
    hipMalloc((void**)&g_edgelist,ne*sizeof(pii));

    //copying host data onto device
    hipMemcpy(g_depth,c_depth,nv*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(g_edgelist,c_edgelist,ne*sizeof(pii),hipMemcpyHostToDevice);
    hipMemcpy(gpu_g,cpu_g,sizeof(graph),hipMemcpyHostToDevice);

    hipMemcpy(&(gpu_g->edgelist),&g_edgelist,sizeof(pii *),hipMemcpyHostToDevice);
    hipMemcpy(&(gpu_g->depth),&g_depth,sizeof(int*),hipMemcpyHostToDevice);

    //invoking kernel to initialize the depth array of the graph 
    init_depth_kernel<<<1,nv>>>(gpu_g,start);

    //declaration of bool variables in host for routine to invoke bfs kernel
    bool *cpu_done;
    cpu_done = (bool*)malloc(sizeof(bool));
    *cpu_done = false;

    //declaration of bool variables in device for routine to invoke bfs kernel   
    bool *gpu_done;
    hipMalloc((void**)&gpu_done,sizeof(bool));
    hipMemcpy(gpu_done,cpu_done,sizeof(bool),hipMemcpyHostToDevice);

    // routine that invokes bfs kernel from host
    simple_bfs(cpu_g,gpu_g,cpu_done,gpu_done);

    //copying device data back into host memory
    hipMemcpy(c_depth,g_depth,nv*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(c_edgelist,g_edgelist,ne*sizeof(pii),hipMemcpyDeviceToHost);
    hipMemcpy(cpu_g,gpu_g,sizeof(graph),hipMemcpyDeviceToHost);

    cpu_g->edgelist = c_edgelist;
    cpu_g->depth = c_depth;

    //printing depth of vertices from host memory
    printgraph(cpu_g);
   
    return 0;

}