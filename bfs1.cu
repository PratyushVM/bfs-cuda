#include<stdio.h>
#include<utility>
#include<hip/hip_runtime.h>
#include<time.h>

class graph
{
public:
    int num_vertices, num_edges;
    int *depth;
    std::pair<int,int> *edgelist;

    graph()
    {
        depth = (int*)malloc(10*sizeof(int));
        // edgelist = (std::pair<int,int> *)malloc(10*sizeof(std::pair<int,int>));
    }

};

void readgraph(graph &g, int argc, char **argv)
{
    int v,e;

    bool fromstdin = (argc > 2);

    if(!fromstdin)
    {
        scanf("%d %d",&v,&e);
    }

    else
    {
        srand(0);
        v = atoi(argv[2]);
        e = atoi(argv[3]);
    }

    g.num_vertices = v;
    g.num_edges = e;

    if(fromstdin)
    {
        int a,b;
        for(int i=0;i<e;i++)
        {
            scanf("%d %d",&a,&b);
            g.edgelist[i] = std::make_pair(a,b);
        }
    }

    else
    {
        int a,b;
        for(int i=0;i<e;i++)
        {
            a = rand() % v;
            b = rand() % v;
            g.edgelist[i] = std::make_pair(a,b);
        }
    }

}

__global__ void initialize(graph g, int start)
{
    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;

    if(id == start)
    {
        g.depth[id] = 0;
    }

    else
    {
        g.depth[id] = -1;
    }

}


__global__ void bfs(graph &g, int start, int current_depth, bool *stop)
{
    unsigned int e = blockIdx.x*blockDim.x + threadIdx.x;

    int v1 = g.edgelist[e].first;
    int v2 = g.edgelist[e].second;

    int d1 = g.depth[v1];
    int d2 = g.depth[v2];

    if(d1 == current_depth && d2 == -1)
    {
        g.depth[v2] = d1 + 1;
        *stop = false;
    }

    else if(d2 == current_depth && d1 == -1)
    {
        g.depth[v1] = d2 + 1;
        *stop = false;
    }

}

int main()
{
    graph g;
    /*int *start = (int*)argv[1];
    int l = 0;

    readgraph(g,argc,argv);
    initialize<<<1,g.num_vertices>>>(g,*start);

    printf("Number of vertices : %d\n",g.num_vertices);
*///    printf("Number of edges : %d\n",g.num_edges);

    hipError_t err = hipSuccess;
    g.num_vertices=5;
    g.num_edges=4;
    for(int i=0;i<5;i++)
    {
        int a = rand() % 5;
        int b = rand() % 5;
        g.edgelist[i] = std::make_pair(a,b);
    }
    int start =2;
    int l=0;
    
    graph *gg = NULL;
    printf("BFS of graph :\n");
    int* a =NULL;
    err = hipMalloc((void**)&a, sizeof(int));
    if (err != hipSuccess)
    {
        printf("failed to allocate mem error: %s",hipGetErrorString(err)); }
    printf("malloc last\n"); //     
    bool *stop;
    
    hipMalloc((void**)&stop,sizeof(bool));
    err = hipMalloc((void**)&gg,sizeof(g));
    if (err != hipSuccess)
    {
        printf("failed to allocate mem error: %s",hipGetErrorString(err)); }
    hipMemcpy(gg,&g,sizeof(g),hipMemcpyHostToDevice);
    
    bool done = false;
    while(!done)
    {
        done = true;
        bfs<<<1,(*gg).num_edges>>>(*gg,start,l,&done);
        hipMemcpy(&done,stop,sizeof(done),hipMemcpyDeviceToHost);
        l++;
    }

    printf("Vertex - Level\n");

    for(int i=0; i<(*gg).num_vertices; i++)
    {
        printf("   %d       %d\n",i,(*gg).depth[i]);
    }

    return 0;
}




 