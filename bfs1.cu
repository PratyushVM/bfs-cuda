#include<stdio.h>
#include<utility>
#include<hip/hip_runtime.h>
#include<time.h>

struct graph
{
    int num_vertices, num_edges;
    int *depth;
    std::pair<int,int> *edgelist;
};

void readgraph(graph &g, int argc, char **argv)
{
    int v,e;

    bool fromstdin = (argc > 2);

    if(!fromstdin)
    {
        scanf("%d %d",&v,&e);
    }

    else
    {
        srand(0);
        v = atoi(argv[2]);
        e = atoi(argv[3]);
    }

    g.num_vertices = v;
    g.num_edges = e;

    if(fromstdin)
    {
        int a,b;
        for(int i=0;i<e;i++)
        {
            scanf("%d %d",&a,&b);
            g.edgelist[i] = std::make_pair(a,b);
        }
    }

    else
    {
        int a,b;
        for(int i=0;i<e;i++)
        {
            a = rand() % v;
            b = rand() % v;
            g.edgelist[i] = std::make_pair(a,b);
        }
    }

}

__global__ void initialize(graph g, int start)
{
    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;

    if(id == start)
    {
        g.depth[id] = 0;
    }

    else
    {
        g.depth[id] = -1;
    }

}


__global__ void bfs(graph &g, int start, int current_depth, bool *stop)
{
    unsigned int e = blockIdx.x*blockDim.x + threadIdx.x;

    int v1 = g.edgelist[e].first;
    int v2 = g.edgelist[e].second;

    int d1 = g.depth[v1];
    int d2 = g.depth[v2];

    if(d1 == current_depth && d2 == -1)
    {
        g.depth[v2] = d1 + 1;
        *stop = false;
    }

    else if(d2 == current_depth && d1 == -1)
    {
        g.depth[v1] = d2 + 1;
        *stop = false;
    }

}

int main(int argc, char **argv)
{
    graph g;
    int *start = (int*)argv[1];
    int l = 0;

    readgraph(g,argc,argv);
    initialize<<<1,g.num_vertices>>>(g,*start);

    printf("Number of vertices : %d\n",g.num_vertices);
    printf("Number of edges : %d\n",g.num_edges);

    graph *gg;
    hipMalloc(&gg,sizeof(g));
    hipMemcpy(gg,&g,sizeof(g),hipMemcpyHostToDevice);

    

    printf("BFS of graph :\n");
    bool done = false;
    bool *stop;
    
    hipMalloc(&stop,sizeof(bool));
    
    while(!done)
    {
        done = true;
        bfs<<<1,(*gg).num_edges>>>(*gg,*start,l,&done);
        hipMemcpy(&done,stop,sizeof(done),hipMemcpyDeviceToHost);
        l++;
    }

    printf("Vertex - Level\n");

    for(int i=0; i<(*gg).num_vertices; i++)
    {
        printf("   %d       %d\n",i,(*gg).depth[i]);
    }

    return 0;
}




 