#include "hip/hip_runtime.h"
#include"include/graph.h"
#include"include/graph.cuh"
#include<bits/stdc++.h>
//main function


int main(int argc, char **argv)
{
    //declaration of variables to store graph data on host and device
    graph *cpu_g,*gpu_g;
    int *c_depth,*g_depth;
    pii *c_edgelist,*g_edgelist;

    //asking user to run with inputs in command line if no inputs are given
    if(argc == 1)
    {
        printf("Enter arguments in command line\n");
        return 0;
    }

    int nv = atoi(argv[1]); //number of vertices
    int ne = atoi(argv[2]); //number of edges
    int start = atoi(argv[3]);  //starting vertex for bfs

    //allocating host memory for data of graph in host
    c_depth = (int*)malloc(nv*sizeof(int));
    c_edgelist = (pii *)malloc(ne*sizeof(pii));

    //invoking function to read graph data from command line
    readgraph(c_edgelist,nv,ne,argc,argv);

    //allocating host memory for graph object
    cpu_g = (graph*)malloc(sizeof(graph));

    //assigning values to data members of graph object from host data
    cpu_g->v = nv;
    cpu_g->e = ne;
    cpu_g->depth = c_depth;
    cpu_g->edgelist = c_edgelist;

    //allocating device memory for graph object on GPU    
    hipMalloc((void**)&gpu_g,sizeof(graph));
    hipMalloc((void**)&g_depth,nv*sizeof(int));
    hipMalloc((void**)&g_edgelist,ne*sizeof(pii));

    //copying host data onto device
    hipMemcpy(g_depth,c_depth,nv*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(g_edgelist,c_edgelist,ne*sizeof(pii),hipMemcpyHostToDevice);
    hipMemcpy(gpu_g,cpu_g,sizeof(graph),hipMemcpyHostToDevice);

    hipMemcpy(&(gpu_g->edgelist),&g_edgelist,sizeof(pii *),hipMemcpyHostToDevice);
    hipMemcpy(&(gpu_g->depth),&g_depth,sizeof(int*),hipMemcpyHostToDevice);

    std::clock_t start_time,end_time;

    start_time = std::clock();

    //invoking kernel to initialize the depth array of the graph 
    init_depth_kernel<<<nblocks,threads_per_block>>>(gpu_g,start);

    //declaration of bool variables in host - used for routine to invoke bfs kernel
    bool *cpu_done;
    cpu_done = (bool*)malloc(sizeof(bool));
    *cpu_done = false;

    //declaration of bool variables in device for routine to invoke bfs kernel   
    bool *gpu_done;
    hipMalloc((void**)&gpu_done,sizeof(bool));
    hipMemcpy(gpu_done,cpu_done,sizeof(bool),hipMemcpyHostToDevice);

    //routine that invokes bfs kernel from host
    simple_bfs(cpu_g,gpu_g,cpu_done,gpu_done,argv);

    end_time = std::clock();

    //copying device data back into host memory
    hipMemcpy(c_depth,g_depth,nv*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(c_edgelist,g_edgelist,ne*sizeof(pii),hipMemcpyDeviceToHost);
    hipMemcpy(cpu_g,gpu_g,sizeof(graph),hipMemcpyDeviceToHost);
    
    cpu_g->edgelist = c_edgelist;
    cpu_g->depth = c_depth;

    //printing depth of vertices from host memory
    //printgraph(cpu_g);

    double time_taken = double(end_time - start_time) / double(CLOCKS_PER_SEC); 
    
    FILE *g = fopen("time.txt","w");
    fprintf(g,"%lf\n",time_taken); 
    printf("Execution time written in time.txt\n");

    check(cpu_g,c_edgelist,start);

    //freeing memory allocated on GPU
    hipFree(gpu_done);
    hipFree(g_depth);
    hipFree(g_edgelist);
    hipFree(gpu_g);

    //freeing memory allocated on CPU
    free(cpu_done);
    free(c_depth);
    free(c_edgelist);
    free(cpu_g);
   
    return 0;

}