#include "hip/hip_runtime.h"
#include"../include/graph.cuh"


//Kernel invoked in simple_bfs routine

__global__ void bfs_kernel(graph *g, bool *done)
{
    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x; //id of thread

    //storing corresponding vertices and depth values into thread's local memory 
    int v1 = g->edgelist[id].f;
    int v2 = g->edgelist[id].s;
    int d1 = g->depth[v1];
    int d2 = g->depth[v2];

    //checking if vertex of next depth value is discovered
    if(d1 != -1 && d2 == -1)
    {
        g->depth[v2] = g->depth[v1] + 1;
        *done = false;
    }
    
    else if(d2 != -1 && d1 == -1)
    {
        g->depth[v1] = g->depth[v2] + 1;
        *done = false;
    }

    //*done is used to determine if the bfs_kernel shoulb be invoked again by simple_bfs routine

}