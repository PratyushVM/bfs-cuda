#include "hip/hip_runtime.h"
#include"../include/graph.cuh"


//Kernel invoked in simple_bfs routine

__global__ void bfs_kernel(graph *g, bool *done, int current_depth)
{
    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x; //id of thread
    if(id < g->e)
    {

        //storing corresponding vertices and depth values into thread's local memory 
        int v1 = g->edgelist[id].first;
        int v2 = g->edgelist[id].second;
        int d1 = g->depth[v1];
        int d2 = g->depth[v2];
        

        //checking if vertex of next depth value is discovered
        if(d1 == current_depth && d2 == -1)
        {
            g->depth[v2] = d1 + 1;
            *done = false;
        }
        
        else if(d2 == current_depth && d1 == -1)
        {
            g->depth[v1] = d2 + 1;
            *done = false;
        }

        //*done is used to determine if the bfs_kernel shoulb be invoked again by simple_bfs routine

    }

}