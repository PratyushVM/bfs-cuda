#include"../include/graph.cuh"

//host function for simple_bfs routine that invokes bfs_kernel iteratively 

void simple_bfs(graph *cpu_g, graph *gpu_g, bool *cpu_done, bool *gpu_done)
{
    while(!(*cpu_done))
    {
        *cpu_done = true;
        hipMemcpy(gpu_done,cpu_done,sizeof(bool),hipMemcpyHostToDevice);
        bfs_kernel<<<1,cpu_g->e>>>(gpu_g,gpu_done);
        hipMemcpy(cpu_done,gpu_done,sizeof(bool),hipMemcpyDeviceToHost);
    }

}