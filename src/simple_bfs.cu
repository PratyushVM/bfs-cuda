#include"../include/graph.cuh"

//host function for simple_bfs routine that invokes bfs_kernel iteratively 

void simple_bfs(graph *cpu_g, graph *gpu_g, bool *cpu_done, bool *gpu_done,char **argv)
{
    int l=0;
    while(!(*cpu_done))
    {
        *cpu_done = true;
        hipMemcpy(gpu_done,cpu_done,sizeof(bool),hipMemcpyHostToDevice);
        bfs_kernel<<<nblocks,threads_per_block>>>(gpu_g,gpu_done,l);
        hipMemcpy(cpu_done,gpu_done,sizeof(bool),hipMemcpyDeviceToHost);
        l++;
    }

}