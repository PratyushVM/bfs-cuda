#include "hip/hip_runtime.h"
#include"../include/graph.cuh"

//Kernel that initializes depth array of graph

__global__ void init_depth_kernel(graph *g, int start)
{
    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x; //id of thread
    if(id<g->v)
    {
        //checking if vertex is the starting point of bfs or not, and initializing depth value respectively
        if(id == start)
        {
            g->depth[id] = 0;
        }

        else
        {
            g->depth[id] = -1;
        }

    }
}